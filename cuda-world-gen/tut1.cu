// Jordan Cazamias
// CUDA World Gen

// tut1.cu: tutorial 1 cude file

#include <iostream>

#include "hip/hip_runtime.h"

using namespace std;

__global__ void AddIntsCUDA(int *a, int *b)
{
	a[0] += b[0];
}

/*
int main()
{
	int a = 5, b = 9;
	int *d_a, *d_b;

	cudaMalloc(&d_a, sizeof(int));
	cudaMalloc(&d_b, sizeof(int));

	cudaMemcpy(d_a, &a, sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(d_b, &b, sizeof(int), cudaMemcpyHostToDevice);

	// Call kernel on GPU
	AddIntsCUDA<<<1, 1>>>(d_a, d_b);

	cudaMemcpy(&a, d_a, sizeof(int), cudaMemcpyDeviceToHost);

	cout << "The answer is " << a << endl;

	cudaFree(d_a);
	cudaFree(d_b);

	return 0;
}
*/