#include "hip/hip_runtime.h"
// Jordan Cazamias
// CUDA World Gen 2015

// Parallel implementation of worldgen_seq

extern "C"
{
	#include "worldgen_pll.cuh"
}

#define CUDA_MAX_BLOCKS 65535

#define CUDA_CALL(x) do {hipError_t status = x; if(status !=hipSuccess) { \
	printf("Error at %s:%d\n",__FILE__, status); \
	exit(status);}} while(0) 
#define CURAND_CALL(x) do {hiprandStatus_t status = x; if(status != HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n",__FILE__, status); \
	exit(status);}} while(0)

/* Function that generates the worldmap */
void GenerateWorldMapPll(unsigned seed, int numFaults);

extern "C" void genworld_pll(int numFaults)
{
	int       NumberOfFaults = 0, a, j, i, Color, MaxZ = 1, MinZ = -1;
	int       row, TwoColorMode = 0;
	int       index2;
	unsigned  Seed = 0;
	int       Threshold, Count;
	int       PercentWater, PercentIce, Cur;
	char SaveName[256];  /* 255 character filenames should be enough? */
	char SaveFile[256];  /* SaveName + .gif */
	FILE * Save;

	// Begin timing for parallel algorithm
	LARGE_INTEGER pll_start, pll_end;
	QueryPerformanceCounter(&pll_start);

	init_worldgen();
	reset_pll_times();

	if (WorldMapArray == NULL)
	{
		WorldMapArray = (int *)malloc(XRange*YRange*sizeof(int));
		if (WorldMapArray == NULL)
		{
			fprintf(stderr, "WorldMapArray could not be allocated.");
			exit(-1);
		}
	}

	if (SinIterPhi == NULL)
	{
		SinIterPhi = (float *)malloc(2 * XRange*sizeof(float));
		if (SinIterPhi == NULL)
		{
			fprintf(stderr, "SinIterPhi could not be allocated.");
			exit(-1);
		}
		for (i = 0; i<XRange; i++)
		{
			SinIterPhi[i] = SinIterPhi[i + XRange] = (float)sin(i * 2 * PI / XRange);
		}
	}

	/*
	fprintf(stderr, "Seed: ");
	scanf("%d", &Seed);
	fprintf(stderr, "Number of faults: ");
	scanf("%d", &NumberOfFaults);
	fprintf(stderr, "Percent water: ");
	scanf("%d", &PercentWater);
	fprintf(stderr, "Percent ice: ");
	scanf("%d", &PercentIce);

	fprintf(stderr, "Save as (.GIF will be appended): ");
	scanf("%8s", SaveName);
	*/

	Seed = time(NULL);
	NumberOfFaults = numFaults;
	PercentWater = 60;
	PercentIce = 10;
	strcpy(SaveName, "default_pll");

	srand(Seed);

	for (j = 0, row = 0; j<XRange; j++)
	{
		WorldMapArray[row] = 0;
		for (i = 1; i<YRange; i++) WorldMapArray[i + row] = INT_MIN;
		row += YRange;
	}

	/* Define some "constants" which we use frequently */
	YRangeDiv2 = (float) YRange / 2;
	YRangeDivPI = (float) YRange / PI;

	/* Generate the map! */
	// Call world generation kernel
	GenerateWorldMapPll(Seed, NumberOfFaults);

	/* Copy data (I have only calculated faults for 1/2 the image.
	* I can do this due to symmetry... :) */
	index2 = (XRange / 2)*YRange;
	for (j = 0, row = 0; j<XRange / 2; j++)
	{
		for (i = 1; i<YRange; i++)                    /* fix */
		{
			WorldMapArray[row + index2 + YRange - i] = WorldMapArray[row + i];
		}
		row += YRange;
	}

	/* Reconstruct the real WorldMap from the WorldMapArray and FaultArray */
	for (j = 0, row = 0; j<XRange; j++)
	{
		/* We have to start somewhere, and the top row was initialized to 0,
		* but it might have changed during the iterations... */
		Color = WorldMapArray[row];
		for (i = 1; i<YRange; i++)
		{
			/* We "fill" all positions with values != INT_MIN with Color */
			Cur = WorldMapArray[row + i];
			if (Cur != INT_MIN)
			{
				Color += Cur;
			}
			WorldMapArray[row + i] = Color;
		}
		row += YRange;
	}

	// Time coloring
	LARGE_INTEGER pll_color_start, pll_color_end;
	QueryPerformanceCounter(&pll_color_start);

	/* Compute MAX and MIN values in WorldMapArray */
	for (j = 0; j<XRange*YRange; j++)
	{
		Color = WorldMapArray[j];
		if (Color > MaxZ) MaxZ = Color;
		if (Color < MinZ) MinZ = Color;
	}

	/* Compute color-histogram of WorldMapArray.
	* This histogram is a very crude aproximation, since all pixels are
	* considered of the same size... I will try to change this in a
	* later version of this program. */
	for (j = 0, row = 0; j<XRange; j++)
	{
		for (i = 0; i<YRange; i++)
		{
			Color = WorldMapArray[row + i];
			Color = (int)(((float)(Color - MinZ + 1) / (float)(MaxZ - MinZ + 1)) * 30) + 1;
			Histogram[Color]++;
		}
		row += YRange;
	}

	/* Threshold now holds how many pixels PercentWater means */
	Threshold = PercentWater*XRange*YRange / 100;

	/* "Integrate" the histogram to decide where to put sea-level */
	for (j = 0, Count = 0; j<256; j++)
	{
		Count += Histogram[j];
		if (Count > Threshold) break;
	}

	/* Threshold now holds where sea-level is */
	Threshold = j*(MaxZ - MinZ + 1) / 30 + MinZ;

	if (TwoColorMode)
	{
		for (j = 0, row = 0; j<XRange; j++)
		{
			for (i = 0; i<YRange; i++)
			{
				Color = WorldMapArray[row + i];
				if (Color < Threshold)
					WorldMapArray[row + i] = 3;
				else
					WorldMapArray[row + i] = 20;
			}
			row += YRange;
		}
	}
	else
	{
		/* Scale WorldMapArray to colorrange in a way that gives you
		* a certain Ocean/Land ratio */
		for (j = 0, row = 0; j<XRange; j++)
		{
			for (i = 0; i<YRange; i++)
			{
				Color = WorldMapArray[row + i];

				if (Color < Threshold)
					Color = (int)(((float)(Color - MinZ) / (float)(Threshold - MinZ)) * 15) + 1;
				else
					Color = (int)(((float)(Color - Threshold) / (float)(MaxZ - Threshold)) * 15) + 16;

				/* Just in case... I DON't want the GIF-saver to flip out! :) */
				if (Color < 1) Color = 1;
				if (Color > 255) Color = 31;
				WorldMapArray[row + i] = Color;
			}
			row += YRange;
		}

		/* "Recycle" Threshold variable, and, eh, the variable still has something
		* like the same meaning... :) */
		Threshold = PercentIce*XRange*YRange / 100;

		if ((Threshold <= 0) || (Threshold > XRange*YRange)) goto Finished;

		FilledPixels = 0;
		/* i==y, j==x */
		for (i = 0; i<YRange; i++)
		{
			for (j = 0, row = 0; j<XRange; j++)
			{
				Color = WorldMapArray[row + i];
				//if (Color < 32) FloodFill4(j, i, Color);
				/* FilledPixels is a global variable which FloodFill4 modifies...
				* I know it's ugly, but as it is now, this is a hack! :)
				*/
				if (FilledPixels > Threshold) goto NorthPoleFinished;
				row += YRange;
			}
		}

	NorthPoleFinished:
		FilledPixels = 0;
		/* i==y, j==x */
		for (i = (YRange - 1); i>0; i--)            /* fix */
		{
			for (j = 0, row = 0; j<XRange; j++)
			{
				Color = WorldMapArray[row + i];
				//if (Color < 32) FloodFill4(j, i, Color);
				/* FilledPixels is a global variable which FloodFill4 modifies...
				* I know it's ugly, but as it is now, this is a hack! :)
				*/
				if (FilledPixels > Threshold) goto Finished;
				row += YRange;
			}
		}
	Finished:;
	}

	// Finish timing coloring
	QueryPerformanceCounter(&pll_color_end);
	pll_color_usec += get_elapsed_usec(pll_color_start, pll_color_end);

	// Start timing save to gif
	LARGE_INTEGER pll_gif_start, pll_gif_end;
	QueryPerformanceCounter(&pll_gif_start);

	/* append .gif to SaveFile */
	sprintf(SaveFile, "%s.gif", SaveName);
	/* open binary SaveFile */
	Save = fopen(SaveFile, "wb");
	/* Write GIF to savefile */

	GIFEncode(Save, XRange, YRange, 1, 0, 8, Red, Green, Blue);

	// Finish timing save to gif
	QueryPerformanceCounter(&pll_gif_end);
	pll_gif_usec += get_elapsed_usec(pll_gif_start, pll_gif_end);

	//fprintf(stderr, "Map created, saved as %s.\n", SaveFile);

	free(WorldMapArray);
	free(SinIterPhi);
	WorldMapArray = NULL;
	SinIterPhi = NULL;

	// Get total algorithm time
	QueryPerformanceCounter(&pll_end);
	pll_total_usec += get_elapsed_usec(pll_start, pll_end);

	return;
}

__global__ void GenCUDA(int *WorldMapArray, float *SinIterPhi, int *XRange, int *YRange, float *rands);


void GenerateWorldMapPll(unsigned seed, int numFaults)
{
	// Determine how many threads should be started
	//int numThreads = (int)XRange / 2;
	int numBlocks = numFaults;
	int threadsPerBlock = (int)XRange / 2;

	int *d_WorldMapArray;
	float *d_SinIterPhi;
	
	// Set up world map array for GPU
	size_t wmaSize = XRange * YRange * sizeof(int);
	//CUDA_CALL(hipMalloc(&d_WorldMapArray, wmaSize));
	CUDA_CALL(hipMalloc(&d_WorldMapArray, wmaSize));
	//printf("WMA last byte: %x\n", WorldMapArray[XRange * YRange - 1]);
	CUDA_CALL(hipMemcpy(d_WorldMapArray, WorldMapArray, wmaSize, hipMemcpyHostToDevice));

	// Set up SinIterPhi for GPU
	size_t sipSize = 2 * XRange * sizeof(float);
	CUDA_CALL(hipMalloc(&d_SinIterPhi, sipSize));
	CUDA_CALL(hipMemcpy(d_SinIterPhi, SinIterPhi, sipSize, hipMemcpyHostToDevice));

	// Set up XRange, YRange
	int *d_XRange;
	int *d_YRange;
	CUDA_CALL(hipMalloc(&d_XRange, sizeof(int)));
	CUDA_CALL(hipMemcpy(d_XRange, &XRange, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CALL(hipMalloc(&d_YRange, sizeof(int)));
	CUDA_CALL(hipMemcpy(d_YRange, &YRange, sizeof(int), hipMemcpyHostToDevice));

	// Begin RNG timing
	LARGE_INTEGER rng_start_time, rng_end_time;
	QueryPerformanceCounter(&rng_start_time);

	// Set up random numbers
	int numRands = 3 * numBlocks;
	float *d_rands;
	CUDA_CALL(hipMalloc(&d_rands, sizeof(float) * numRands));

	// Create pseudo-random number generator
	hiprandGenerator_t gen;
	//CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT)); 
	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT)); 
	// Set ordering (for increased performance)
	CURAND_CALL(hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED));
	// Set seed
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed)); 
	// Generate n floats on device
	CURAND_CALL(hiprandGenerateUniform(gen, d_rands, numRands));

	// End RNG timing
	QueryPerformanceCounter(&rng_end_time);
	pll_rng_usec += get_elapsed_usec(rng_start_time, rng_end_time);

	// Begin Comp timing
	LARGE_INTEGER comp_start, comp_end;
	QueryPerformanceCounter(&comp_start);

	// ***** Call kernel ******
	int curBlock;
	int runBlocks = min(numBlocks, CUDA_MAX_BLOCKS);
	for (curBlock = 0; curBlock < numBlocks; curBlock += runBlocks)
	{
		GenCUDA<<<runBlocks, threadsPerBlock>>>(d_WorldMapArray, d_SinIterPhi, d_XRange, d_YRange, d_rands);
	}
	hipError_t status = hipDeviceSynchronize();

	// End Comp timing
	QueryPerformanceCounter(&comp_end);
	pll_comp_usec += get_elapsed_usec(comp_start, comp_end);

	// Retrieve world map array data from GPU
	CUDA_CALL(hipMemcpy(WorldMapArray, d_WorldMapArray, wmaSize, hipMemcpyDeviceToHost));

	CUDA_CALL(hipFree(d_WorldMapArray));
	CUDA_CALL(hipFree(d_SinIterPhi));
}

__global__ void GenCUDA(int *WorldMapArray, float *SinIterPhi, int *XRange, int *YRange, float *rands)
{
	__shared__ float		 Alpha;
	__shared__ float		 Beta;
	__shared__ float         TanB;
	__shared__ int		     Xsi;
	__shared__ unsigned int  flag1;
	int			  *wma_ptr;
	int			  Phi;
	int			  Theta;

	// Calculate which Phi thread should take care of
	//int idx = threadIdx.x + (blockIdx.x * blockDim.x);
	Phi = threadIdx.x;
	//printf("Phi = %d + (%d * %d)\n", threadIdx.x, blockIdx.x, blockDim.x);
	//printf("Thread id: (%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);

	if (Phi == 0)
	{
		// Extract random values
		//float rand[3];
		//rand[0] = rands[blockIdx.x * 3];
		//rand[1] = rands[blockIdx.x * 3 + 1];
		//rand[2] = rands[blockIdx.x * 3 + 2];

		flag1 = (int)(rands[blockIdx.x * 3] + 0.5);

		/* Create a random greatcircle...
		* Start with an equator and rotate it */
		Alpha = (rands[blockIdx.x * 3 + 1] - 0.5)*PI; /* Rotate around x-axis */
		Beta = (rands[blockIdx.x * 3 + 2] - 0.5)*PI; /* Rotate around y-axis */
		//printf("(flag1, Alpha, Beta): (%u, %f, %f)\n", flag1, Alpha, Beta);

		TanB = tan(acos(cos(Alpha)*cos(Beta)));
		Xsi = (int)((*XRange) / 2 - ((*XRange) / PI) * Beta);
		//printf("Xsi: %d\n", Xsi);
	}

	__syncthreads();
	//printf("XRange, YRange: %d, %d\n", *XRange, *YRange);

	//for (Phi = 0; Phi < XRange / 2; Phi++)
	//{
		//float YRangeDivPI = (*YRange) / PI;
		//float YRangeDiv2 = (*YRange) / 2;
		//printf("pll (siniterphi, sin) = (%f, %f)\n", SinIterPhi[Xsi - Phi + (*XRange)], sin((Xsi - Phi) * 2 * PI / (*XRange)));
		//int row = (*YRange) * Phi;
		//printf("pll_row: %d\n", row);
		Theta = (int)(((*YRange) / 2) * atan(SinIterPhi[Xsi - Phi + (*XRange)] * TanB)) + ((*YRange) / 2);
		//printf("Phi, sip, theta: %d, %f, %d\n", Phi, SinIterPhi[Xsi - Phi + (*XRange)], Theta);
		wma_ptr = WorldMapArray + ((*YRange) * Phi + Theta);

		atomicCAS(wma_ptr, INT_MIN, 0);
		if (flag1)
		{
			/* Rise northen hemisphere <=> lower southern */
			atomicSub(wma_ptr, 1);
		}
		else
		{
			/* Rise southern hemisphere */
			atomicAdd(wma_ptr, 1);
		}
	//}
}
